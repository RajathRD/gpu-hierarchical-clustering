/* 
 * This file contains the code for doing the heat distribution problem. 
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s), memory allocation, data movement, etc. 
 * 
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <float.h>
#include <math.h>

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)
#define PRINT_LOG 0
#define PRINT_ANALYSIS 0
/* Define constants */
#define RANGE 100

/*****************************************************************/

// Function declarations
void  seq_clustering(float *, unsigned int, unsigned int, int *, float *);
//void  gpu_clustering(float *, unsigned int, unsigned int);
void calculate_pairwise_dists(float *, int, int, float *);
void find_pairwise_min(float *, int, float *, int *);
void merge_clusters(int *, int, int, int);
float calculate_dist(float *, int, int, int);
void print_float_matrix(float *, int, int);
void print_int_matrix(int *, int, int);
int get_parent(int, int *);
// Kernel functions
//__global__ void calculateMatrix(float * temp_d, float * playground_d, unsigned int N);

// Helper functions
void print_float_matrix(float * a, int n, int m){
  for(int i=0; i<n; i++){
    for(int j=0; j<m; j++)
      printf("%f ", a[index(i, j, m)]);
    printf("\n");
  }
}

void print_int_matrix(int * a, int n, int m){
  for (int i=0; i<n; i++){
    for(int j=0; j<m; j++)
      printf("%d ", a[index(i,j,m)]);
    printf("\n");
  }
}

void load_data(float * dataset, int n, int m) {
  srand((unsigned int) 0);
  for (int i = 0; i < n; i ++) {
    for (int j = 0; j < m; j++) {
      // assign numbers between 0 and RANGE
      dataset[index(i, j, m)] = ((float)rand()/(float)(RAND_MAX)) * RANGE - RANGE/2.0;
    } 
  }
  if (PRINT_LOG){
    printf("Dataset:\n");
    print_float_matrix(dataset, n, m);
  }
}


/*****************************************************************/
/**** Do NOT CHANGE ANYTHING in main() function ******/

int main(int argc, char * argv[])
{
  //Define variables
  //unsigned int N; /* Dimention of NxN matrix */
  int type_of_device = atoi(argv[3]); // CPU or GPU
  int n = atoi(argv[1]);
  int m = atoi(argv[2]);

  printf("Hierarchical Clustering:\n");
  printf("Dataset size: %d x %d\n", n, m);
  printf("Device Type: %d\n", type_of_device);
  
  // to measure time taken by a specific part of the code 
  double time_taken;
  clock_t start, end;
  
  // Validate
  /*if(argc != 4)
  {
    fprintf(stderr, "usage: heatdist num  iterations  who\n");
    fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
    fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
    fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU execution\n");
    exit(1);
  }*/

  //Load data
  float * dataset;
  dataset = (float *)calloc(n*m, sizeof(float));
  if( !dataset )
  {
   fprintf(stderr, " Cannot allocate the %u x %u array\n", n, m);
   exit(1);
  }
  load_data(dataset, n, m);
  printf("Data loaded!\n");
  
  type_of_device = atoi(argv[3]);

  //N = (unsigned int) atoi(argv[1]);
  //iterations = (unsigned int) atoi(argv[2]);
 
  
  /* Dynamically allocate NxN array of floats */
  /*playground = (float *)calloc(N*N, sizeof(float));
  if( !playground )
  {
   fprintf(stderr, " Cannot allocate the %u x %u array\n", N, N);
   exit(1);
  }*/
  
  /* Initialize it: calloc already initalized everything to 0 */
  // Edge elements  initialization
  /*for(i = 0; i < N; i++)
    playground[index(0,i,N)] = 100;
  // FIXME: Why N-1? Shouldnt it be N? There is a post about it in Brightspace which has not been answered yet.
  // Will leave it as it is
  for(i = 0; i < N-1; i++)
    playground[index(N-1,i,N)] = 150;
  */
  float dendrogram[(n-1)*3];
  int * result;
  result = (int *)calloc(n, sizeof(int));
  if( !type_of_device ) // The CPU sequential version
  {  
    start = clock();
    seq_clustering(dataset, n, m, result, dendrogram);    
    end = clock();
  }
  else  // The GPU version
  {
     start = clock();
     end = clock();    
  }
  
  
  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  
  printf("Time taken for %s is %lf\n", type_of_device == 0? "CPU" : "GPU", time_taken);

  free(dataset);
  free(result);

  return 0;

}


/*****************  The CPU sequential version (DO NOT CHANGE THAT) **************/
void  seq_clustering(float * dataset, unsigned int n, unsigned int m, int* result, float * dendrogram)
{
  // to measure time taken by a specific part of the code 
  double time_taken;
  clock_t start, end;

  /* Dynamically allocate another array for temp values */
  /* Dynamically allocate NxN array of floats */
  
  if( !result )
  {
   fprintf(stderr, " Cannot allocate result %u array\n", n);
   exit(1);
  }
  for (int i = 0; i < n; i++) result[i] = i;

  float* dist_matrix = (float *)calloc(n*n, sizeof(float));
  if( !dist_matrix )
  {
   fprintf(stderr, " Cannot allocate dist_matrix %u array\n", n*n);
   exit(1);
  }

  // O(n*n*m) -> GPU
  start = clock();
  calculate_pairwise_dists(dataset, n, m, dist_matrix);
  end = clock();

  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  if (PRINT_ANALYSIS)
    printf("Time taken for distance computation: %lf\n", time_taken);
  
  for (int iteration=0; iteration < n - 1; iteration++) {
    
    float entry[3]; 
    // O(I*n*n) -> GPU
    start = clock();
    find_pairwise_min(dist_matrix, n, entry, result);
    end = clock();
    time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
    if (PRINT_ANALYSIS)
      printf("Time taken for pairwise min, Iteration %d: %lf\n", iteration, time_taken);
    dendrogram[index(iteration, 0, 3)] = entry[0];
    dendrogram[index(iteration, 1, 3)] = entry[1];
    dendrogram[index(iteration, 2, 3)] = entry[2];
    // O(I*n) -> amortized O(I)
    start = clock();
    merge_clusters(result, (int)entry[0], (int)entry[1], n);
    end = clock();
    time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
    if (PRINT_ANALYSIS)
      printf("Time taken for merge cluster, Iteration %d: %lf\n", iteration, time_taken);
    if (PRINT_LOG){
      printf("Iteartion #%d\n", iteration);
      printf("Min Indices: %d, %d\n", (int)entry[0], (int)entry[1]);
      print_int_matrix(result, 1, n);
    }
    
  }

  for (int i=0; i<n; i++) result[i] = get_parent(i, result);

  if (PRINT_LOG){
    printf("Cluster IDs:\n");
    print_int_matrix(result, 1, n);
    printf("Dendrogram:\n");
    print_float_matrix(dendrogram, n-1, 3);
  }

  free(dist_matrix);
  //num_bytes = N*N*sizeof(float);
  /* Copy initial array in temp */
  //memcpy((void *)temp, (void *) playground, num_bytes);
  /* Move new values into old values */ 
  //memcpy((void *)playground, (void *) temp, num_bytes);
}

void calculate_pairwise_dists(float * dataset, int n, int m, float * dist_matrix) {
  // O(n)
  // for (int i = 0; i < n*n; i++) dist_matrix[i] = FLT_MAX;
  
  // O(n*n*m)
  for (int i = 0; i < n; i++) {
    for (int j = i+1; j < n; j++) {
      // O(m)
      dist_matrix[index(i, j, n)] = calculate_dist(dataset, i, j, m);
    }
  }  
}

// passing vec1_i and vec2_i instead of float * as dist_matrix is 1-D
float calculate_dist(float * dataset, int i, int j, int dim) {
  float dist = 0;
  // O(m)
  for (int mi = 0; mi < dim; mi++){
    dist += (dataset[index(i, mi, dim)] - dataset[index(j,mi,dim)]) * (dataset[index(i, mi, dim)] - dataset[index(j,mi,dim)]);
  }
  return dist;
}


int get_parent(int curr_parent, int* parents) {
  if (parents[curr_parent] == curr_parent) return curr_parent;
  parents[curr_parent] = get_parent(parents[curr_parent], parents);
  return parents[curr_parent];
  // return get_parent(parents[curr_parent], parents);
}


void find_pairwise_min(float * dist_matrix, int n, float* entry, int* parents) {
  entry[0] = 0;
  entry[1] = 0;
  entry[2] = FLT_MAX;
  for (int i = 0; i < n; i++) {
    for (int j = i+1; j < n; j++) {
      if (get_parent(i, parents) != get_parent(j, parents)){
      // if (parents[i] != parents[j]) {
        float curr_dist = dist_matrix[index(i, j, n)];
        if (curr_dist < entry[2]) {
          entry[0] = i;
          entry[1] = j;
          entry[2] = curr_dist;
        }
      }
    }
  }
}


void merge_clusters(int * result, int data_point_i, int data_point_j, int dim) {
  if (!(data_point_i >= 0 && data_point_i < dim && data_point_j >= 0 && data_point_j < dim)) {
    printf("merge_clusters out of bounds");
    return;
  } 
  // int cluster_j = result[data_point_j];
  // for(int i=0; i<dim; i++)
  //   if(result[i] == cluster_j)
  //     result[i] = result[data_point_i];
  int parent_i = get_parent(data_point_i, result);
  result[get_parent(data_point_j, result)] = parent_i;
} 

/***************** The GPU version: Write your code here *********************/
/* This function can call one or more kernels if you want ********************/
/*void  gpu_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{

  size_t num_bytes = N*N*sizeof(float);
  float * playground_d;

  // Move data to device memory
  cudaMalloc((void**) &playground_d, num_bytes);
  cudaMemcpy(playground_d, playground, num_bytes, cudaMemcpyHostToDevice);

  // Maximum number of threads per block in cuda1.cims.nyu.edu 
  int thread_cnt = 1024;
  int block_cnt = (int) ceil((double) N*N / thread_cnt);

  float * temp_d;
  cudaMalloc((void**) &temp_d, num_bytes);
  cudaMemcpy(temp_d, playground_d, num_bytes, cudaMemcpyDeviceToDevice);
  for (int k = 0; k < iterations; k++) 
  {           
    calculateMatrix<<<block_cnt, thread_cnt>>>(temp_d, playground_d, N);
    cudaMemcpy(playground_d, temp_d, num_bytes, cudaMemcpyDeviceToDevice); 
  }

  // Move new values into old values
  cudaMemcpy(playground, temp_d, num_bytes, cudaMemcpyDeviceToHost);
  cudaFree(playground_d);
  cudaFree(temp_d);
}

__global__ void calculateMatrix(float * temp_d, float * playground_d, unsigned int N)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  // Dont update if thread id is outside of the box
  if (idx >= N * N) return;
  int i = idx / N;
  int j = idx % N;
  
  // Dont update the edges
  if (i-1 >= 0 && i+1 < N && j-1 >= 0 && j+1 < N)
  {
    temp_d[index(i,j,N)] = (playground_d[index(i-1,j,N)] +
                            playground_d[index(i+1,j,N)] +
                            playground_d[index(i,j-1,N)] +
                            playground_d[index(i,j+1,N)])/4.0;
  }
} 
*/

/* Helper Functions */
/*void print_matrix(float * matrix, unsigned int N)
{
  for (int i = 0; i < N; i++) 
  {
    for (int j = 0; j < N; j++) 
    {
      float curr = matrix[index(i,j,N)];
      printf("%.2f\t", curr);
    }
    printf("\n");
  }
}
*/

